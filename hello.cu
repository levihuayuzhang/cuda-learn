
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void multi(int *a, int *b, int *res){
    *res = (*a) * (*b);
}

int main(void) {
    int a, b, c; // host copies
    int *d_a, *d_b, *d_c; // device copies
    int size = sizeof(int);

    // allocate memory for device copies
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    a= 3;
    b= 4;

    // copy host intput value to device
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    multi<<<1,1>>>(d_a, d_b, d_c);

    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    printf("Hello world! Result is %d", c);

    return 0;
}