#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <hip/hip_runtime.h>
#include "common.h"
#include <chrono>

int recursiveReduce(int *data, int const size)
{
    if (size == 1) return data[0];
    int const stride = size/2;
    
    for (int i = 0; i < stride; i++)
    {
        data[i] += data[i + stride];
    }

    return recursiveReduce(data, stride);
}

/**
 * @brief 
 * Kernel 1
 * Neighbored pair implementation with divergence
 * only use even number of thread, highly divergent
 * 
 * @param g_idata global input data address
 * @param d_odata global output data address
 * @param n data set size
 *
 */
__global__ void reduceNeighbored (int *g_idata, int *g_odata, unsigned int n)
{
    unsigned int tid = threadIdx.x;
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // data index for each bloack (local pointer) in global mem
	int *idata = g_idata + blockIdx.x * blockDim.x;

    // boundary check
	if (idx >= n) return;

	for (int stride = 1; stride < blockDim.x; stride *= 2)
	{
		if ((tid % (2 * stride)) == 0)
		{
			idata[tid] += idata[tid+stride];
		}
		__syncthreads(); // inner block sync (wait for each block to finish)
	}

    // store result from each block (1st elements) to out put array
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

/**
 * @brief kernel 2 
 * less active thread compared to kernel 1
 * 
 * @param g_idata 
 * @param g_odata 
 * @param n 
 * @return __global__ 
 */
__global__ void reduceNeighboredLess (int *g_idata, int *g_odata, unsigned int n)
{
    unsigned int tid = threadIdx.x;
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // data index for each bloack (local pointer) in global mem
	int *idata = g_idata + blockIdx.x * blockDim.x;

    // boundary check
	if (idx >= n) return;

	for (int stride = 1; stride > 0; stride *= 2)
	{   // array index for 
        int index = 2 * stride * tid;

		if (index < blockDim.x)
		{
            // use index
			idata[index] += idata[index + stride];
		}
		__syncthreads(); // inner block sync (wait for each block to finish)
	}
    

    // store result from each block (1st elements) to out put array
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

/**
 * @brief kernel 3
 * 
 * @param g_idata 
 * @param g_odata 
 * @param n 
 * @return __global__ 
 */
__global__ void reduceInterLeaved (int *g_idata, int *g_odata, unsigned int n)
{
    unsigned int tid = threadIdx.x;
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // data index for each bloack (local pointer) in global mem
	int *idata = g_idata + blockIdx.x * blockDim.x;

    // boundary check
	if (idx >= n) return;

    for (int stride = blockDim.x / 2; stride < blockDim.x; stride >> 1)
	{    
		if (tid < stride)
		{
			idata[tid] += idata[tid+ stride];
		}
		__syncthreads(); // inner block sync (wait for each block to finish)
	} 


    // store result from each block (1st elements) to out put array
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}

int main(int argc, char const *argv[])
{
    int dev = 0;
    hipDeviceProp_t devicePro;
    hipGetDeviceProperties(&devicePro, dev);
    std::cout << argv[0] << "starting reduction at " << std::endl;
    std::cout << "device " << dev << ": " << devicePro.name << std::endl;

    bool bRsult = false;
    int size  = 1 << 24;
    std::cout << "With arry size of " << size << std::endl;
    int blockSize = 512;
    
    if (argc > 1) blockSize = atoi(argv[1]);

    dim3 block (blockSize, 1);
	dim3 grid ((size + block.x - 1) / block.x, 1);
    std::cout << "Grid  " << grid.x << " Block " << block.x << std::endl;
	
    // host mem
	size_t bytes = size * sizeof(int);
    int *h_idata = (int *) malloc(bytes);
    int *h_odata = (int *) malloc(grid.x * sizeof(int));
    int *tmp = (int *) malloc(bytes);

    // initialize random data
    for (int i = 0; i < size; i++)
    {
        h_idata[i] = (int)(rand() *0xFF);
    }    
    memcpy(tmp, h_idata, bytes);

    // double iStart, iElaps;
    // device mem
    int *d_idata = NULL;
    int *d_odata = NULL;
    CHECK(hipMalloc((void **) &d_idata, bytes));
    CHECK(hipMalloc((void **) &d_odata, grid.x * sizeof(int)));

    // cpu reduction
    auto iStart = std::chrono::high_resolution_clock::now();
    int cpu_sum = recursiveReduce(tmp, size);
    auto endTime = std::chrono::high_resolution_clock::now();
    auto iElaps = std::chrono::duration_cast<std::chrono::microseconds>(endTime-iStart);
    double cpu_duration = iElaps.count();
    std::cout << "\nCPU reduce recusice: " << cpu_duration << " microsec cpu_sum: " 
                << cpu_sum << std::endl;

    int gpu_sum;
    // kernel 1: reduceNeighboared
    hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    iStart = std::chrono::high_resolution_clock::now();

    reduceNeighbored<<<grid, block>>>(d_idata, d_odata, size);
    hipDeviceSynchronize();

    endTime = std::chrono::high_resolution_clock::now();
    iElaps = std::chrono::duration_cast<std::chrono::microseconds>(endTime-iStart);
    double gpu_duration = iElaps.count();
    double improved = (cpu_duration - gpu_duration) / cpu_duration * 100;

    hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i = 0; i < grid.x; i++) gpu_sum += h_odata[i];

    std::cout << "GPU reduce neighbored: " << gpu_duration << " microsec gpu_sum: " 
                << gpu_sum << " with "<< improved << "% improved!" <<std::endl;
    if (gpu_sum != cpu_sum) std::cout << "Test result failed: sum result not match!" << std::endl;


    // kernel 2: reduceNeighboaredLess
    hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    iStart = std::chrono::high_resolution_clock::now();

    reduceNeighboredLess<<<grid, block>>>(d_idata, d_odata, size);
    hipDeviceSynchronize();

    endTime = std::chrono::high_resolution_clock::now();
    iElaps = std::chrono::duration_cast<std::chrono::microseconds>(endTime-iStart);
    gpu_duration = iElaps.count();
    improved = (cpu_duration - gpu_duration) / cpu_duration * 100;
    
    hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i = 0; i < grid.x; i++) gpu_sum += h_odata[i];

    std::cout << "GPU reduce neighbored less: " << gpu_duration << " microsec gpu_sum: " 
                << gpu_sum << " with "<< improved << "% improved!" <<std::endl;
    if (gpu_sum != cpu_sum) std::cout << "Test result failed: sum result not match!" << std::endl; 

    // kernel 3: interleaved 
    hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    iStart = std::chrono::high_resolution_clock::now();

    reduceInterLeaved<<<grid, block>>>(d_idata, d_odata, size);
    hipDeviceSynchronize();

    endTime = std::chrono::high_resolution_clock::now();
    iElaps = std::chrono::duration_cast<std::chrono::microseconds>(endTime-iStart);
    gpu_duration = iElaps.count();
    improved = (cpu_duration - gpu_duration) / cpu_duration * 100;
    
    hipMemcpy(h_odata, d_odata, grid.x * sizeof(int), hipMemcpyDeviceToHost);
    gpu_sum = 0;
    for (int i = 0; i < grid.x; i++) gpu_sum += h_odata[i];

    std::cout << "GPU reduce interleaved: " << gpu_duration << " microsec gpu_sum: " 
                << gpu_sum << " with "<< improved << "% improved!" <<std::endl;
    if (gpu_sum != cpu_sum) std::cout << "Test result failed: sum result not match!" << std::endl; 




    
    
    
    
    
    
    
    // clean resource
    free(h_idata); free(h_odata);
    hipFree(d_idata); hipFree(d_odata);
    hipDeviceReset();
    

    std::cout << "end!!!" << std::endl;
    return 0;
}
