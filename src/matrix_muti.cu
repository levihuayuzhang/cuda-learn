/**
 * @file matrix_muti.cu
 * @author Huayu Zhang (zhanghuayu.dev@gmail.com)
 * 
 * @brief 
 * Matrix multi
 * 
 * Reference: https://zhuanlan.zhihu.com/p/34587739?utm_id=0
 * @version 0.1
 * @date 2023-10-29
 * 
 */
#include <iostream>
// #include <cuda.h>
#include <hip/hip_runtime.h>

struct Matrix
{
    int width;
    int height;
    float *elements;
};

__device__ float getElement(Matrix *A, int row, int col) {
    return A->elements[row * A->width + col];
}

__device__ void setElement(Matrix *A, int row, int col, float value) {
    A->elements[row * A->width + col] = value;
}

__global__ void matrixMulti(Matrix *A, Matrix *B, Matrix *C) {
    float CVal = 0.0; 

    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int column = threadIdx.x + blockIdx.x * blockDim.x;

    for (int i = 0; i < A->width; ++i) {
        CVal += getElement(A, row, i) * getElement(B, i, column);
    }

    setElement(C, row, column, CVal);
}


int main (void) {
    int width = 1<<10;
    int height = 1<<10;
    Matrix *A, *B, *C;

    hipMallocManaged((void**)&A, sizeof(Matrix));
    hipMallocManaged((void**)&B, sizeof(Matrix));
    hipMallocManaged((void**)&C, sizeof(Matrix));

    int nBytes = width*height*sizeof(float);
    hipMallocManaged((void**)&A->elements, nBytes);
    hipMallocManaged((void**)&B->elements, nBytes);
    hipMallocManaged((void**)&C->elements, nBytes);

    A->height = height;
    A->width = width;
    B->height = height;
    B->width = width;
    C->height = height;
    C->width = width;
    for (int i = 0; i < width * height; ++i)
    {
        A->elements[i] = 1.0;
        B->elements[i] = 2.0;
    }

    dim3 blockSize(32, 32);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, 
                    (height + blockSize.y - 1) / blockSize.y);

    matrixMulti<<<gridSize, blockSize>>>(A, B, C);

    hipDeviceSynchronize();

    float maxError= 0.0;
    for (int i = 0; i < width*height; ++i) {
        maxError = fmax(maxError, fabs(C->elements[i] - 2*width));
    }
    std::cout<< "Max Error is: " << maxError << std::endl;

    return 0;
}