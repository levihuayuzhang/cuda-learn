#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipblas.h>

// void random_floats(float* a, int count) {
//     for (int i=0; i < count; i++){
//         a[i] = rand() % 1000;
//     }
// }

__global__ void saxpy_cuda(int n, float a, float *x, float *y) {
    int t_id = threadIdx.x + blockIdx.x * blockDim.x; // thread index
    int stride = blockDim.x * gridDim.x;
    for (int i = t_id; i < n; i+= stride) {
        y[i] = x[i] + y[i];
    }
}

int main(void) {
    const int N = 1UL << 25;
    int size = N * sizeof(float);

    float *x, *y, alpha=2.0;

    // random_floats(x, N);
    // random_floats(y, N);
    hipMalloc(&x, size);
    hipMalloc(&y, size);

    saxpy_cuda<<<32, 1024>>>(N, alpha, x, y);
    hipDeviceSynchronize();

    hipFree(x);
    hipFree(y);


}
